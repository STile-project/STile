#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cmath>
#include "spmm_utils/dense_tile.h"
#include "spmm_utils/sparse_tile.h"
#include "spmm_utils/compute_utils.h"
#include "spmm_utils/output_tile.h"
#include <stdio.h>
#include <mma.h>

using namespace nvcuda;

//4-bit Tile_N = 64 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_4b_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    // Each int32 has 8 4-bit integers with double buffers
    __shared__ int values_tile_array[Tile_K*VecLength/4];
    __shared__ int column_indices_tile_array[Tile_K*2];

    //padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has eight 4-bit integers
    wmmaSparseTile_4b<LoadType, VecType, Tile_K * VecLength / 8, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_4b<Tile_K * VecLength / 8> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_4b output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix, scale);
    output_tile_storer.Store();
}

//8-bit Tile_N = 64 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_8b_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has four 8-bit integers
    wmmaSparseTile_8b<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_8b<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix, scale);
    output_tile_storer.Store();
}

//16-bit 8-bit Tile_N = 64 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_16b8b_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b8b<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b8b output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix, scale);
    output_tile_storer.Store();
}


//16-bit 8-bit Tile_N = 64 with 2 warps 8v
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_16b8b8v_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b8v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b8b8v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b8b8v output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix, scale);
    output_tile_storer.Store();
}

//8-bit A 4-bit B Tile_N = 64 warps = 2
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_8b4b_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_8b4b<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    //wmmaComputeUtils_8b4b<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment, lane_id);
    wmmaComputeUtils_4b<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b4b output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix, scale);
    output_tile_storer.Store();
}


//8-bit A 4-bit B Tile_N = 64 warps = 2, 8v
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__device__ void wmmaSpmm_kernel_8b4b8v_(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_8b4b8v<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_8b4b8v<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b4b8v output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix, scale);
    output_tile_storer.Store();
}


//4-bit Tile_N = 64 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_4b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_4b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

//4-bit Tile_N = 64 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_4b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_4b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}


//8-bit Tile_N = 64 with 2 waprs
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_8b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_8b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_8b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}


//16-bit 8-bit Tile_N = 64 with 2 waprs
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b8b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_16b8b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_16b8b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_16b8b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

//16-bit 8-bit Tile_N = 64 with 2 waprs 8v
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b8b8v(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_16b8b8v_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_16b8b8v(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_16b8b8v_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

//8-bit A 4-bit B Tile_N = 64 warps = 2
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b4b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_8b4b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}

template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_8b4b(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_8b4b_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}


//8-bit A 4-bit B Tile_N = 64 warps = 2, 8v
template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b4b8v(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    wmmaSpmm_kernel_8b4b8v_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}


template <typename LoadType, typename IndexType, typename VecType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void batched_wmmaSpmm_kernel_8b4b8v(
    int m_vec, int dimN, int dimK, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    int entry_idx = blockIdx.z;
    const VecType* values = values_b + entry_idx * values_stride;
    const int* rhs_matrix = rhs_matrix_b + entry_idx * rhs_stride;
    half* output_matrix = output_matrix_b + entry_idx * output_stride;

    wmmaSpmm_kernel_8b4b8v_<LoadType, IndexType, VecType, Tile_K, Tile_N, Warps, VecLength>(
    m_vec, dimN, dimK, scale,
    row_indices, 
    row_offsets,
    column_indices,
    values,
    rhs_matrix,
    output_matrix);
}



template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_4b_template(
    int m_vec, int vec_length, int n, int k, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    wmmaSpmm_kernel_4b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);

    return hipGetLastError();
}


template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_8b_template(
    int m_vec, int vec_length, int n, int k, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);

    wmmaSpmm_kernel_8b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

//8-bit 4-bit Tile_N = 64 with 2 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_8b4b_template(
    int m_vec, int vec_length, int n, int k, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_8b4b8v<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    else
        wmmaSpmm_kernel_8b4b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

//16-bit 8-bit Tile_N = 64 with 2 waprs
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_16b8b_template(
    int m_vec, int vec_length, int n, int k, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    half* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_16b8b8v<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    else
        wmmaSpmm_kernel_16b8b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}


template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t batched_wmmaSpmm_4b_template(
    int m_vec, int vec_length, int n, int k, int batch_size, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), batch_size);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    batched_wmmaSpmm_kernel_4b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);

    return hipGetLastError();
}


template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t batched_wmmaSpmm_8b_template(
    int m_vec, int vec_length, int n, int k, int batch_size, float scale, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), batch_size);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);

    batched_wmmaSpmm_kernel_8b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);
    return hipGetLastError();
}

//8-bit 4-bit Tile_N = 64 with 2 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t batched_wmmaSpmm_8b4b_template(
    int m_vec, int vec_length, int n, int k, int batch_size, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), batch_size);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        batched_wmmaSpmm_kernel_8b4b8v<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);
    else
        batched_wmmaSpmm_kernel_8b4b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);
    return hipGetLastError();
}

//16-bit 8-bit Tile_N = 64 with 2 waprs
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t batched_wmmaSpmm_16b8b_template(
    int m_vec, int vec_length, int n, int k, int batch_size, float scale,
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values_b,
    int values_stride,
    const int* __restrict__ rhs_matrix_b,
    int rhs_stride,
    half* __restrict__ output_matrix_b,
    int output_stride)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), batch_size);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        batched_wmmaSpmm_kernel_16b8b8v<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);
    else
        batched_wmmaSpmm_kernel_16b8b<int, int, VecType, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, scale, row_indices, row_offsets, column_indices, values_b, values_stride, rhs_matrix_b, rhs_stride, output_matrix_b, output_stride);
    return hipGetLastError();
}


torch::Tensor batched_deq_spmm_mma_16b8b(
    torch::Tensor row_indices,
    torch::Tensor row_offsets,
    torch::Tensor column_indices,
    torch::Tensor values,
    torch::Tensor rhs_matrix,
    int vec_length,
    int bits_lhs,
    int bits_rhs,
    float scale)
{
    int rhs_num_items_per_int32 = 32 / bits_rhs;

    int m_vec = row_offsets.size(-1)/2;
    int m = m_vec * vec_length;

    int n_int32 = rhs_matrix.size(-1);

    int n = n_int32 * rhs_num_items_per_int32;

    int k = rhs_matrix.size(-2);

    //int batch_size = rhs_matrix.numel() / (n * k);
    int batch_size = rhs_matrix.size(-3);

    int nnz = column_indices.numel();

    int values_stride = 0;
    if(vec_length == 8)
	values_stride = nnz * 2; //2xlong long for 16b8v
    else
	values_stride = nnz;
    int rhs_stride = k * n_int32;
    int output_stride = m * n;

    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(rhs_matrix.device());

    auto output_matrix = torch::empty({batch_size, m, n}, options);


    switch(vec_length){
        case 2:
            batched_wmmaSpmm_16b8b_template<int, int, 1, 16, 64, 32, 2, 2>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<int *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 4:
            batched_wmmaSpmm_16b8b_template<int, long long, 1, 16, 64, 32, 2, 4>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<long long *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 8:
            batched_wmmaSpmm_16b8b_template<int, long long, 1, 16, 64, 32, 2, 8>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<long long *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }
    return output_matrix;
}



torch::Tensor batched_deq_spmm_mma_4b(
    torch::Tensor row_indices,
    torch::Tensor row_offsets,
    torch::Tensor column_indices,
    torch::Tensor values,
    torch::Tensor rhs_matrix,
    int vec_length,
    int bits_lhs,
    int bits_rhs,
    float scale)
{
    int rhs_num_items_per_int32 = 32 / bits_rhs;

    int m_vec = row_offsets.size(-1)/2;
    int m = m_vec * vec_length;

    int n_int32 = rhs_matrix.size(-1);

    int n = n_int32 * rhs_num_items_per_int32;

    int k = rhs_matrix.size(-2);

    //int batch_size = rhs_matrix.numel() / (n * k);
    int batch_size = rhs_matrix.size(-3);

    int nnz = column_indices.numel();


    int values_stride = nnz; //stride in vector format
    int rhs_stride = k * n_int32;
    int output_stride = m * n;

    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(rhs_matrix.device());

    auto output_matrix = torch::empty({batch_size, m, n}, options);


    switch(vec_length){
        case 2:
            batched_wmmaSpmm_4b_template<int, char, 1, 32, 64, 32, 2, 2>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<char *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 4:
            batched_wmmaSpmm_4b_template<int, short, 1, 32, 64, 32, 2, 4>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<short *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 8:
            batched_wmmaSpmm_4b_template<int, int, 1, 32, 64, 32, 2, 8>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<int *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }
    return output_matrix;

}



torch::Tensor batched_deq_spmm_mma_8b4b(
    torch::Tensor row_indices,
    torch::Tensor row_offsets,
    torch::Tensor column_indices,
    torch::Tensor values,
    torch::Tensor rhs_matrix,
    int vec_length,
    int bits_lhs,
    int bits_rhs,
    float scale)
{
    int rhs_num_items_per_int32 = 32 / bits_rhs;

    int m_vec = row_offsets.size(-1)/2;
    int m = m_vec * vec_length;

    int n_int32 = rhs_matrix.size(-1);

    int n = n_int32 * rhs_num_items_per_int32;

    int k = rhs_matrix.size(-2);

    //int batch_size = rhs_matrix.numel() / (n * k);
    int batch_size = rhs_matrix.size(-3);

    int nnz = column_indices.numel();


    int values_stride = nnz; //stride in vector format
    int rhs_stride = k * n_int32;
    int output_stride = m * n;

    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(rhs_matrix.device());

    auto output_matrix = torch::empty({batch_size, m, n}, options);


    switch(vec_length){
        case 2:
            batched_wmmaSpmm_8b4b_template<int, short, 1, 32, 64, 32, 2, 2>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<short *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 4:
            batched_wmmaSpmm_8b4b_template<int, int, 1, 32, 64, 32, 2, 4>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<int *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 8:
            batched_wmmaSpmm_8b4b_template<int, long long, 1, 32, 64, 32, 2, 8>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<long long *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }
    return output_matrix;
}

torch::Tensor batched_deq_spmm_mma_8b(
    torch::Tensor row_indices,
    torch::Tensor row_offsets,
    torch::Tensor column_indices,
    torch::Tensor values,
    torch::Tensor rhs_matrix,
    int vec_length,
    int bits_lhs,
    int bits_rhs,
    float scale)
{
    //int lhs_num_items_per_int32 = 32 / bits_lhs;
    int rhs_num_items_per_int32 = 32 / bits_rhs;

    int m_vec = row_offsets.size(-1)/2;
    int m = m_vec * vec_length;

    int n_int32 = rhs_matrix.size(-1);

    int n = n_int32 * rhs_num_items_per_int32;

    int k = rhs_matrix.size(-2);

    //int batch_size = rhs_matrix.numel() / (n * k);
    int batch_size = rhs_matrix.size(-3);

    int nnz = column_indices.numel();


    int values_stride = nnz; //stride in vector format
    int rhs_stride = k * n_int32;
    int output_stride = m * n;

    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(rhs_matrix.device());

    auto output_matrix = torch::empty({batch_size, m, n}, options);


    switch(vec_length){
        case 2:
            batched_wmmaSpmm_8b_template<int, short, 1, 16, 64, 32, 2, 2>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<short *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 4:
            batched_wmmaSpmm_8b_template<int, int, 1, 16, 64, 32, 2, 4>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<int *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        case 8:
            batched_wmmaSpmm_8b_template<int, long long, 1, 16, 64, 32, 2, 8>(m_vec, vec_length, n, k, batch_size, scale,
                row_indices.data<int>(), row_offsets.data<int>(), column_indices.data<int>(), 
                reinterpret_cast<long long *>(values.data<int>()),
                values_stride,
                reinterpret_cast<int *>(rhs_matrix.data<int>()),
                rhs_stride, 
                reinterpret_cast<half *>(output_matrix.data<torch::Half>()),
	        output_stride);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }
    return output_matrix;

}
